#include "hip/hip_runtime.h"
//#include "GenerateProposalConfigs.h"
#include "GenerateProposalLayer.h"

using namespace GenerateProposal;
/*

static Logger gLogger;
using namespace nvinfer1;
using namespace nvcaffeparser1;
using namespace plugin;

// stuff we know about the network and the caffe input/output blobs
static const int INPUT_C = 3;
static const int INPUT_H = 375;
static const int INPUT_W = 500;
static const int IM_INFO_SIZE = 3;
static const int OUTPUT_CLS_SIZE = 21;
static const int OUTPUT_BBOX_SIZE = OUTPUT_CLS_SIZE * 4;
static int gUseDLACore{-1};

const std::string CLASSES[OUTPUT_CLS_SIZE]{"background", "aeroplane", "bicycle", "bird", "boat", "bottle", "bus", "car", "cat", "chair", "cow", "diningtable", "dog", "horse", "motorbike", "person", "pottedplant", "sheep", "sofa", "train", "tvmonitor"};

const char* INPUT_BLOB_NAME0 = "data";
const char* INPUT_BLOB_NAME1 = "im_info";
const char* OUTPUT_BLOB_NAME0 = "bbox_pred";
const char* OUTPUT_BLOB_NAME1 = "cls_prob";
const char* OUTPUT_BLOB_NAME2 = "rois";
*/
namespace nvinfer1{
    GenerateProposalLayerPlugin::GenerateProposalLayerPlugin(const int cudaThread /*= 512*/):mThreadCount(cudaThread){
        /*mClassCount = CLASS_NUM;
        mGenerateProposalKernel.clear();
        mGenerateProposalKernel.push_back(yolo1);
        mGenerateProposalKernel.push_back(yolo2);
        mGenerateProposalKernel.push_back(yolo3);

        mKernelCount = mGenerateProposalKernel.size();*/
    }
    GenerateProposalLayerPlugin::~GenerateProposalLayerPlugin(){
        if(mInputBuffer)
            CUDA_CHECK(hipHostFree(mInputBuffer));
        if(mOutputBuffer)
            CUDA_CHECK(hipHostFree(mOutputBuffer));
    }
    // create the plugin at runtime from a byte stream
    GenerateProposalLayerPlugin::GenerateProposalLayerPlugin(const void* data, size_t length){
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mKernelCount);
        mGenerateProposalKernel.resize(mKernelCount);
        auto kernelSize = mKernelCount*sizeof(GenerateProposalKernel);
        memcpy(mGenerateProposalKernel.data(),d,kernelSize);
        d += kernelSize;

        assert(d == a + length);
    }

    void GenerateProposalLayerPlugin::serialize(void* buffer)
    {
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mKernelCount);
        auto kernelSize = mKernelCount*sizeof(GenerateProposalKernel);
        memcpy(d,mGenerateProposalKernel.data(),kernelSize);
        d += kernelSize;

        assert(d == a + getSerializationSize());
    }
    
    size_t GenerateProposalLayerPlugin::getSerializationSize()
    {  
        return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount) + sizeof(GenerateProposal::GenerateProposalKernel) * mGenerateProposalKernel.size();
    }

    int GenerateProposalLayerPlugin::initialize()
    {
            /*
            int totalCount = 0;
            for(const auto& yolo : mGenerateProposalKernel)
                totalCount += (LOCATIONS + 1 + mClassCount) * yolo.width*yolo.height * CHECK_COUNT;
            CUDA_CHECK(hipHostAlloc(&mInputBuffer, totalCount * sizeof(float), hipHostMallocDefault));

            totalCount = 0;//detection count
            for(const auto& yolo : mGenerateProposalKernel)
                totalCount += yolo.width*yolo.height * CHECK_COUNT;
            CUDA_CHECK(hipHostAlloc(&mOutputBuffer, sizeof(float) + totalCount * sizeof(Detection), hipHostMallocDefault));
            */
            /*
            void doInference(IExecutionContext& context, float* inputData, float* inputImInfo, float* outputBboxPred, float* outputClsProb, float* outputRois, int batchSize)
            {
                const ICudaEngine& engine = context.getEngine();
                // input and output buffer pointers that we pass to the engine - the engine requires exactly IEngine::getNbBindings(),
                // of these, but in this case we know that there is exactly 2 inputs and 3 outputs.
                assert(engine.getNbBindings() == 5);
                /////////////////////// moved below //////////////////
                context.enqueue(batchSize, buffers, stream, nullptr);
                CHECK(hipMemcpyAsync(outputBboxPred, buffers[outputIndex0], batchSize * nmsMaxOut * OUTPUT_BBOX_SIZE * sizeof(float), hipMemcpyDeviceToHost, stream));
                CHECK(hipMemcpyAsync(outputClsProb, buffers[outputIndex1], batchSize * nmsMaxOut * OUTPUT_CLS_SIZE * sizeof(float), hipMemcpyDeviceToHost, stream));
                CHECK(hipMemcpyAsync(outputRois, buffers[outputIndex2], batchSize * nmsMaxOut * 4 * sizeof(float), hipMemcpyDeviceToHost, stream));
                hipStreamSynchronize(stream);

                // release the stream and the buffers
                hipStreamDestroy(stream);
                CHECK(hipFree(buffers[inputIndex0]));
                CHECK(hipFree(buffers[inputIndex1]));
                CHECK(hipFree(buffers[outputIndex0]));
                CHECK(hipFree(buffers[outputIndex1]));
                CHECK(hipFree(buffers[outputIndex2]));
            }
            */
            void* buffers[5];

            // In order to bind the buffers, we need to know the names of the input and output tensors.
            // note that indices are guaranteed to be less than IEngine::getNbBindings()
            int inputIndex0 = engine.getBindingIndex(INPUT_BLOB_NAME0),
                inputIndex1 = engine.getBindingIndex(INPUT_BLOB_NAME1),
                outputIndex0 = engine.getBindingIndex(OUTPUT_BLOB_NAME0),
                outputIndex1 = engine.getBindingIndex(OUTPUT_BLOB_NAME1),
                outputIndex2 = engine.getBindingIndex(OUTPUT_BLOB_NAME2);

            // create GPU buffers and a stream
            CHECK(hipMalloc(&buffers[inputIndex0], batchSize * INPUT_C * INPUT_H * INPUT_W * sizeof(float)));   // data
            CHECK(hipMalloc(&buffers[inputIndex1], batchSize * IM_INFO_SIZE * sizeof(float)));                  // im_info
            CHECK(hipMalloc(&buffers[outputIndex0], batchSize * nmsMaxOut * OUTPUT_BBOX_SIZE * sizeof(float))); // bbox_pred
            CHECK(hipMalloc(&buffers[outputIndex1], batchSize * nmsMaxOut * OUTPUT_CLS_SIZE * sizeof(float)));  // cls_prob
            CHECK(hipMalloc(&buffers[outputIndex2], batchSize * nmsMaxOut * 4 * sizeof(float)));                // rois

            hipStream_t stream;
            CHECK(hipStreamCreate(&stream));

            // DMA the input to the GPU,  execute the batch asynchronously, and DMA it back:
            CHECK(hipMemcpyAsync(buffers[inputIndex0], inputData, batchSize * INPUT_C * INPUT_H * INPUT_W * sizeof(float), hipMemcpyHostToDevice, stream));
            CHECK(hipMemcpyAsync(buffers[inputIndex1], inputImInfo, batchSize * IM_INFO_SIZE * sizeof(float), hipMemcpyHostToDevice, stream));
            return 0;
    }
    
    Dims GenerateProposalLayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
            //output the result to channel
            int totalCount = 0;
            for(const auto& yolo : mGenerateProposalKernel)
                totalCount += yolo.width*yolo.height * CHECK_COUNT * sizeof(Detection) / sizeof(float);

            return Dims3(totalCount + 1, 1, 1);
    }

    void GenerateProposalLayerPlugin::forwardCpu(const float*const * inputs, float* outputs, hipStream_t stream)
    {
            auto Logist = [=](float data){
                return 1./(1. + exp(-data));
            };

            CUDA_CHECK(hipStreamSynchronize(stream));
            int i = 0;
            float* inputData = (float *)mInputBuffer; 
            for(const auto& yolo : mGenerateProposalKernel)
            {
                int size = (LOCATIONS + 1 + mClassCount) * yolo.width*yolo.height * CHECK_COUNT;
                CUDA_CHECK(hipMemcpyAsync(inputData, inputs[i], size * sizeof(float), hipMemcpyDeviceToHost, stream));
                inputData += size;
                ++ i;
            }

            inputData = (float *)mInputBuffer;
            std::vector <Detection> result;
            for (const auto& yolo : mGenerateProposalKernel)
            {
                int stride = yolo.width*yolo.height;
                for (int j = 0;j < stride ;++j)
                {
                    for (int k = 0;k < CHECK_COUNT; ++k )
                    {
                        int beginIdx = (LOCATIONS + 1 + mClassCount)* stride *k + j;
                        int objIndex = beginIdx + LOCATIONS*stride;
                        
                        //check obj
                        float objProb = Logist(inputData[objIndex]);   
                        if(objProb <= IGNORE_THRESH)
                            continue;

                        //classes
                        int classId = -1;
                        float maxProb = IGNORE_THRESH;
                        for (int c = 0;c< mClassCount;++c){
                            float cProb =  Logist(inputData[beginIdx + (5 + c) * stride]) * objProb;
                            if(cProb > maxProb){
                                maxProb = cProb;
                                classId = c;
                            }
                        }
            
                        if(classId >= 0) {
                            Detection det;
                            int row = j / yolo.width;
                            int cols = j % yolo.width;
    
                            //Location
                            det.bbox[0] = (cols + Logist(inputData[beginIdx]))/ yolo.width;
                            det.bbox[1] = (row + Logist(inputData[beginIdx+stride]))/ yolo.height;
                            det.bbox[2] = exp(inputData[beginIdx+2*stride]) * yolo.anchors[2*k];
                            det.bbox[3] = exp(inputData[beginIdx+3*stride]) * yolo.anchors[2*k + 1];
                            det.classId = classId;
                            det.prob = maxProb;
                            //det.objectness = objProb;

                            result.emplace_back(det);
                        }
                    }
                }

                inputData += (LOCATIONS + 1 + mClassCount) * stride * CHECK_COUNT;
            }

            
            int detCount =result.size();
            auto data = (float *)mOutputBuffer;
            //copy count;
            data[0] = (float)detCount;
            //std::cout << "detCount"<< detCount << std::endl;
            data++;
            //copy result
            memcpy(data,result.data(),result.size()*sizeof(Detection));

            //(count + det result)
            CUDA_CHECK(hipMemcpyAsync(outputs, mOutputBuffer, sizeof(float) + result.size()*sizeof(Detection), hipMemcpyHostToDevice, stream));
    };

    __device__ float Logist(float data){ return 1./(1. + exp(-data)); };

    __global__ void CalDetection(const float *input, float *output,int noElements, 
            int yoloWidth,int yoloHeight,const float anchors[CHECK_COUNT*2],int classes) {
 
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements) return;

        int stride = yoloWidth*yoloHeight;

        for (int k = 0;k < CHECK_COUNT; ++k )
        {
            int beginIdx = (LOCATIONS + 1 + classes)* stride *k + idx;
            int objIndex = beginIdx + LOCATIONS*stride;
            
            //check objectness
            float objProb = Logist(input[objIndex]);   
            if(objProb <= IGNORE_THRESH)
                continue;

            int row = idx / yoloWidth;
            int cols = idx % yoloWidth;
            
            //classes
            int classId = -1;
            float maxProb = IGNORE_THRESH;
            for (int c = 0;c<classes;++c){
                float cProb =  Logist(input[beginIdx + (5 + c) * stride]) * objProb;
                if(cProb > maxProb){
                    maxProb = cProb;
                    classId = c;
                }
            }

            if(classId >= 0) {
                int resCount = (int)atomicAdd(output,1);
                char* data = (char * )output + sizeof(float) + resCount*sizeof(Detection);
                Detection* det =  (Detection*)(data);

                //Location
                det->bbox[0] = (cols + Logist(input[beginIdx]))/ yoloWidth;
                det->bbox[1] = (row + Logist(input[beginIdx+stride]))/ yoloHeight;
                det->bbox[2] = exp(input[beginIdx+2*stride]) * anchors[2*k];
                det->bbox[3] = exp(input[beginIdx+3*stride]) * anchors[2*k + 1];
                det->classId = classId;
                det->prob = maxProb;
            }
        }
    }
   
    void GenerateProposalLayerPlugin::forwardGpu(const float *const * inputs,float * output,hipStream_t stream) {
        int numElem;
        void* devAnchor;
        size_t AnchorLen = sizeof(float)* CHECK_COUNT*2;
        CUDA_CHECK(hipMalloc(&devAnchor,AnchorLen));

        //first detect count init 0
        CUDA_CHECK(hipMemset(output, 0, sizeof(float)));
        for (unsigned int i = 0;i< mGenerateProposalKernel.size();++i)
        {
            const auto& yolo = mGenerateProposalKernel[i];
            numElem = yolo.width*yolo.height;

            //copy anchor to device
	        CUDA_CHECK(hipMemcpy(devAnchor,yolo.anchors,AnchorLen,hipMemcpyHostToDevice));

            CalDetection<<< (yolo.width*yolo.height + mThreadCount - 1) / mThreadCount, mThreadCount>>>
                    (inputs[i],output, numElem, yolo.width, yolo.height, (float *)devAnchor, mClassCount);
        }
        CUDA_CHECK(hipFree(devAnchor));
    }


    int GenerateProposalLayerPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        assert(batchSize == 1);
        
        //GPU
        forwardGpu((const float *const *)inputs,(float *)outputs[0],stream);

        //CPU
        //forwardCpu((const float *const *)inputs,(float *)outputs[0],stream);
        return 0;
    };

}
/*

TEST(GenerateProposalsTest, TestRealDownSampledGPU) {
  if (!HasCudaGPU())
    return;
  Workspace ws;
  OperatorDef def;
  def.set_name("test");
  def.set_type("GenerateProposals");
  def.add_input("scores");
  def.add_input("bbox_deltas");
  def.add_input("im_info");
  def.add_input("anchors");
  def.add_output("rois");
  def.add_output("rois_probs");
  def.mutable_device_option()->set_device_type(PROTO_CUDA);
  const int img_count = 2;
  const int A = 2;
  const int H = 4;
  const int W = 5;

  vector<float> scores{
      5.44218998e-03f, 1.19207997e-03f, 1.12379994e-03f, 1.17181998e-03f,
      1.20544003e-03f, 6.17993006e-04f, 1.05261997e-05f, 8.91025957e-06f,
      9.29536981e-09f, 6.09605013e-05f, 4.72735002e-04f, 1.13482002e-10f,
      1.50015003e-05f, 4.45032993e-06f, 3.21612994e-08f, 8.02662980e-04f,
      1.40488002e-04f, 3.12508007e-07f, 3.02616991e-06f, 1.97759000e-08f,
      2.66913995e-02f, 5.26766013e-03f, 5.05053019e-03f, 5.62100019e-03f,
      5.37420018e-03f, 5.26280981e-03f, 2.48894998e-04f, 1.06842002e-04f,
      3.92931997e-06f, 1.79388002e-03f, 4.79440019e-03f, 3.41609990e-07f,
      5.20430971e-04f, 3.34090000e-05f, 2.19159006e-07f, 2.28786003e-03f,
      5.16703985e-05f, 4.04523007e-06f, 1.79227004e-06f, 5.32449000e-08f};
  vector<float> bbx{
      -1.65040009e-02f, -1.84051003e-02f, -1.85930002e-02f, -2.08263006e-02f,
      -1.83814000e-02f, -2.89172009e-02f, -3.89706008e-02f, -7.52277970e-02f,
      -1.54091999e-01f, -2.55433004e-02f, -1.77490003e-02f, -1.10340998e-01f,
      -4.20190990e-02f, -2.71421000e-02f, 6.89801015e-03f,  5.71171008e-02f,
      -1.75665006e-01f, 2.30021998e-02f,  3.08554992e-02f,  -1.39333997e-02f,
      3.40579003e-01f,  3.91070992e-01f,  3.91624004e-01f,  3.92527014e-01f,
      3.91445011e-01f,  3.79328012e-01f,  4.26631987e-01f,  3.64892989e-01f,
      2.76894987e-01f,  5.13985991e-01f,  3.79999995e-01f,  1.80457994e-01f,
      4.37402993e-01f,  4.18545991e-01f,  2.51549989e-01f,  4.48318988e-01f,
      1.68564007e-01f,  4.65440989e-01f,  4.21891987e-01f,  4.45928007e-01f,
      3.27155995e-03f,  3.71480011e-03f,  3.60032008e-03f,  4.27092984e-03f,
      3.74579988e-03f,  5.95752988e-03f,  -3.14473989e-03f, 3.52022005e-03f,
      -1.88564006e-02f, 1.65188999e-03f,  1.73791999e-03f,  -3.56074013e-02f,
      -1.66615995e-04f, 3.14146001e-03f,  -1.11830998e-02f, -5.35363983e-03f,
      6.49790000e-03f,  -9.27671045e-03f, -2.83346009e-02f, -1.61233004e-02f,
      -2.15505004e-01f, -2.19910994e-01f, -2.20872998e-01f, -2.12831005e-01f,
      -2.19145000e-01f, -2.27687001e-01f, -3.43973994e-01f, -2.75869995e-01f,
      -3.19516987e-01f, -2.50418007e-01f, -2.48537004e-01f, -5.08224010e-01f,
      -2.28724003e-01f, -2.82402009e-01f, -3.75815988e-01f, -2.86352992e-01f,
      -5.28333001e-02f, -4.43836004e-01f, -4.55134988e-01f, -4.34897989e-01f,
      -5.65053988e-03f, -9.25739005e-04f, -1.06790999e-03f, -2.37016007e-03f,
      -9.71166010e-04f, -8.90910998e-03f, -1.17592998e-02f, -2.08992008e-02f,
      -4.94231991e-02f, 6.63906988e-03f,  3.20469006e-03f,  -6.44695014e-02f,
      -3.11607006e-03f, 2.02738005e-03f,  1.48096997e-02f,  4.39785011e-02f,
      -8.28424022e-02f, 3.62076014e-02f,  2.71668993e-02f,  1.38250999e-02f,
      6.76669031e-02f,  1.03252999e-01f,  1.03255004e-01f,  9.89722982e-02f,
      1.03646003e-01f,  4.79663983e-02f,  1.11014001e-01f,  9.31736007e-02f,
      1.15768999e-01f,  1.04014002e-01f,  -8.90677981e-03f, 1.13103002e-01f,
      1.33085996e-01f,  1.25405997e-01f,  1.50051996e-01f,  -1.13038003e-01f,
      7.01059997e-02f,  1.79651007e-01f,  1.41055003e-01f,  1.62841007e-01f,
      -1.00247003e-02f, -8.17587040e-03f, -8.32176022e-03f, -8.90108012e-03f,
      -8.13035015e-03f, -1.77263003e-02f, -3.69572006e-02f, -3.51580009e-02f,
      -5.92143014e-02f, -1.80795006e-02f, -5.46086021e-03f, -4.10550982e-02f,
      -1.83081999e-02f, -2.15411000e-02f, -1.17953997e-02f, 3.33894007e-02f,
      -5.29635996e-02f, -6.97528012e-03f, -3.15250992e-03f, -3.27355005e-02f,
      1.29676998e-01f,  1.16080999e-01f,  1.15947001e-01f,  1.21797003e-01f,
      1.16089001e-01f,  1.44875005e-01f,  1.15617000e-01f,  1.31586999e-01f,
      1.74735002e-02f,  1.21973999e-01f,  1.31596997e-01f,  2.48907991e-02f,
      6.18605018e-02f,  1.12855002e-01f,  -6.99798986e-02f, 9.58312973e-02f,
      1.53593004e-01f,  -8.75087008e-02f, -4.92327996e-02f, -3.32239009e-02f};
  vector<float> im_info{60, 80, 0.166667f};
  vector<float> anchors{-38, -16, 53, 31, -120, -120, 135, 135};

  // Doubling everything related to images, to simulate
  // num_images = 2
  scores.insert(scores.begin(), scores.begin(), scores.end());
  bbx.insert(bbx.begin(), bbx.begin(), bbx.end());
  im_info.insert(im_info.begin(), im_info.begin(), im_info.end());

  ERMatXf rois_gt(18, 5);
  rois_gt << 0, 0, 0, 79, 59, 0, 0, 5.0005703f, 51.6324f, 42.6950f, 0,
      24.13628387f, 7.51243401f, 79, 45.0663f, 0, 0, 7.50924301f, 67.4779f,
      45.0336, 0, 0, 23.09477997f, 50.61448669f, 59, 0, 0, 39.52141571f,
      51.44710541f, 59, 0, 23.57396317f, 29.98791885f, 79, 59, 0, 0,
      41.90219116f, 79, 59, 0, 0, 23.30098343f, 78.2413f, 58.7287f, 1, 0, 0, 79,
      59, 1, 0, 5.0005703f, 51.6324f, 42.6950f, 1, 24.13628387f, 7.51243401f,
      79, 45.0663f, 1, 0, 7.50924301f, 67.4779f, 45.0336, 1, 0, 23.09477997f,
      50.61448669f, 59, 1, 0, 39.52141571f, 51.44710541f, 59, 1, 23.57396317f,
      29.98791885f, 79, 59, 1, 0, 41.90219116f, 79, 59, 1, 0, 23.30098343f,
      78.2413f, 58.7287f;

  vector<float> rois_probs_gt{2.66913995e-02f,
                              5.44218998e-03f,
                              1.20544003e-03f,
                              1.19207997e-03f,
                              6.17993006e-04f,
                              4.72735002e-04f,
                              6.09605013e-05f,
                              1.50015003e-05f,
                              8.91025957e-06f};

  // Doubling everything related to images, to simulate
  // num_images = 2
  rois_probs_gt.insert(
      rois_probs_gt.begin(), rois_probs_gt.begin(), rois_probs_gt.end());

  AddInput<HIPContext>(
      vector<int64_t>{img_count, A, H, W}, scores, "scores", &ws);
  AddInput<HIPContext>(
      vector<int64_t>{img_count, 4 * A, H, W}, bbx, "bbox_deltas", &ws);
  AddInput<HIPContext>(vector<int64_t>{img_count, 3}, im_info, "im_info", &ws);
  AddInput<HIPContext>(vector<int64_t>{A, 4}, anchors, "anchors", &ws);

  def.add_arg()->CopyFrom(MakeArgument("spatial_scale", 1.0f / 16.0f));
  def.add_arg()->CopyFrom(MakeArgument("pre_nms_topN", 6000));
  def.add_arg()->CopyFrom(MakeArgument("post_nms_topN", 300));
  def.add_arg()->CopyFrom(MakeArgument("nms_thresh", 0.7f));
  def.add_arg()->CopyFrom(MakeArgument("min_size", 16.0f));
  def.add_arg()->CopyFrom(MakeArgument("correct_transform_coords", true));

  unique_ptr<OperatorBase> op(CreateOperator(def, &ws));
  EXPECT_NE(nullptr, op.get());
  EXPECT_TRUE(op->Run());

  // test rois
  Blob* rois_blob = ws.GetBlob("rois");
  EXPECT_NE(nullptr, rois_blob);
  auto& rois_gpu = rois_blob->Get<TensorCUDA>();
  Tensor rois{CPU};
  rois.CopyFrom(rois_gpu);

  EXPECT_EQ(rois.sizes(), (vector<int64_t>{rois_gt.rows(), rois_gt.cols()}));
  auto rois_data =
      Eigen::Map<const ERMatXf>(rois.data<float>(), rois.dim(0), rois.dim(1));
  EXPECT_NEAR((rois_data.matrix() - rois_gt).cwiseAbs().maxCoeff(), 0, 1e-4);

  // test rois_probs
  Blob* rois_probs_blob = ws.GetBlob("rois_probs");
  EXPECT_NE(nullptr, rois_probs_blob);
  auto& rois_probs_gpu = rois_probs_blob->Get<TensorCUDA>();
  Tensor rois_probs{CPU};
  rois_probs.CopyFrom(rois_probs_gpu);
  EXPECT_EQ(
      rois_probs.sizes(), (vector<int64_t>{int64_t(rois_probs_gt.size())}));
  auto rois_probs_data =
      ConstEigenVectorArrayMap<float>(rois_probs.data<float>(), rois.dim(0));
  EXPECT_NEAR(
      (rois_probs_data.matrix() - utils::AsEArrXt(rois_probs_gt).matrix())
          .cwiseAbs()
          .maxCoeff(),
      0,
      1e-4);
}  

*/
